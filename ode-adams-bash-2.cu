#include "hip/hip_runtime.h"
#define NO_STDIO
#include "spherical.h"

static REAL *diffx;
static REAL *olddiffx;
static REAL *tempx;
static int first = 1;

/* x_vec = a_vec + b*c_vec */
__global__ void madd(REAL *x_vec, REAL *a_vec, REAL b, REAL *c_vec, int length) {
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  if (i<length)
    x_vec[i] = a_vec[i] + b*c_vec[i];
  __syncthreads();
}

/* x_vec += b*c_vec */
__global__ void maddto(REAL *x_vec, REAL b, REAL *c_vec, int length) {
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  if (i<length)
    x_vec[i] += b*c_vec[i];
  __syncthreads();
}

/* x_vec += b*c_vec+d*e_vec */
/*
__global__ void maddto_twice(REAL *x_vec, REAL b, REAL *c_vec, REAL d, REAL *e_vec, int length) {
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  if (i<length)
    x_vec[i] += b*c_vec[i] + d*e_vec[i];
  __syncthreads();
}
*/

void ode_adams_bash_2_solve(REAL *t, REAL *x, REAL h, int do_many,
                            param_list_t *param, param_list_t *param_d) {
  if (first) {
    first = 0;
    hipMalloc((void**)&diffx,sizeof(REAL)*param->length);
    hipMalloc((void**)&tempx,sizeof(REAL)*param->length);
    hipMalloc((void**)&olddiffx,sizeof(REAL)*param->length);
/* Midpoint method */
//    derivs(*t,x,diffx,param,param_d);
    compute_psidot(diffx,x,param,param_d,0,1);
    hipMemcpy(olddiffx,diffx,param->length*sizeof(REAL),hipMemcpyDeviceToDevice);
    madd<<<param->length/64+1,64>>>(tempx,x,h/2,diffx, param->length);
//    derivs(*t+h/2,tempx,diffx,param,param_d);
    compute_psidot(diffx,tempx,param,param_d,0,1);
    maddto<<<param->length/64+1,64>>>(x,h,diffx, param->length);
    hipMemcpy(diffx,olddiffx,param->length*sizeof(REAL),hipMemcpyDeviceToDevice);
  } else {
/* Adams-Bashforth method of order 2 */
//    derivs(*t,x,diffx,param,param_d);
    if (do_many)
      compute_psidot(diffx,x,param,param_d,1/*do_adams_bash_2*/,param->print_every);
    else
      compute_psidot(diffx,x,param,param_d,1/*do_adams_bash_2*/,1);
//    maddto_twice<<<param->length/64+1,64>>>(x,3*h/2,diffx,-h/2,olddiffx, param->length);
  }
  if (do_many)
    *t += param->print_every*h;
  else
    *t += h;
}
