#include "hip/hip_runtime.h"
/*
 * Created from psidot.conf by expand-iterate.pl.
 */

#define NO_STDIO
#include "spherical.h"

#define block_width 8

static int first_mc_0 = 1;
static REAL *mult_constant_0;
#define mc_count_0 45
static void initialize_mc_0(param_list_t *param);

/* Declarations of threading go here. */
static int first_thread_0 = 1;
REAL *mult_constant_0_d;
texture<float,1,hipReadModeElementType> mult_constant_0_r;

#define mc_0(i) mult_constant_0[((i)*param->data_width/2+l/2)*param->data_width+m]

//#define mc_0_r(i) tex1Dfetch(mult_constant_0_r,((i)*data_width/2+l/2)*data_width+m)

#define mc_0_r(i) mult_constant_0[((i)*data_width/2+l/2)*data_width+m]

#define promote_to_float2(x) (*((float2*)(&(x))))

__global__ void cuda_thread_0(REAL* psidot_io, REAL* psi_in, param_list_t *param, REAL* mult_constant_0, int do_adams_bash_2) {
  __shared__ REAL lambda;
  __shared__ REAL Dr;
  __shared__ REAL gamm[9], w[3];
#define index(psi,ll,mm,c) psi[(((ll)-l_base+PADDING)/2*(block_width+2*PADDING)+(mm)-m_base+PADDING)*2+c]
  __shared__ REAL psi[(block_width+PADDING)*(block_width+2*PADDING)*2];
  int l = 2*(blockIdx.y*blockDim.y + threadIdx.y);
  int m = blockIdx.x*blockDim.x + threadIdx.x;
  int l_base = 2*(blockIdx.y*blockDim.y);
  int m_base = blockIdx.x*blockDim.x;
/*if (threadIdx.y&(block_width/2)) l_half = PADDING; else l_half = -PADDING;
  if (threadIdx.x&(block_width/2)) m_half = PADDING; else m_half = -PADDING; */
  int l_half = ((threadIdx.y&(block_width/2))-(block_width/4))/(block_width/4)*PADDING;
  int m_half = ((threadIdx.x&(block_width/2))-(block_width/4))/(block_width/4)*PADDING;
  REAL psidot[2];
  int job_nr = threadIdx.y*block_width+threadIdx.x;
  int data_width = param->data_width;
  REAL h = param->h;
#undef ind
#define ind(l,m,c) ind_macro(l,m,c,data_width)
  float2 olddiffx;

  if (m_base<=l_base+block_width) {
    if (job_nr==0)
      lambda = param->lambda;
    if (job_nr==1)
      Dr = param->Dr;
    if (0<=job_nr-2 && job_nr-2<9)
      gamm[job_nr-2] = param->gamm[job_nr-2];
    if (0<=job_nr-11 && job_nr-11<3)
      w[job_nr-11] = param->w[job_nr-11];
    promote_to_float2(index(psi,l,m,0)) = promote_to_float2(psi_in[ind(l,m,0)]);
    promote_to_float2(index(psi,l,m+m_half,0)) = promote_to_float2(psi_in[ind(l,m+m_half,0)]);
    promote_to_float2(index(psi,l+l_half,m,0)) = promote_to_float2(psi_in[ind(l+l_half,m,0)]);
    promote_to_float2(index(psi,l+l_half,m+m_half,0)) = promote_to_float2(psi_in[ind(l+l_half,m+m_half,0)]);
/*
 * Condon-Shortley phase:
 * Y_l^(-m) = (-1)^m conj(Y_l^m)
 */
    if (m<=PADDING) {
      index(psi,l,-m,0) = (m&1) ? -index(psi,l,m,0) : index(psi,l,m,0);
      index(psi,l,-m,1) = !(m&1) ? -index(psi,l,m,1) : index(psi,l,m,1);
    }
//    promote_to_float2(psidot[0]) = promote_to_float2(psidot_io[ind(l,m,0)]);
  }
  __syncthreads();
  if (l<=param->max_order && m<=l) {
      psidot[0] = w[0]*((-mc_0_r(0))*index(psi,l,m-1,1)+(-mc_0_r(1))*index(psi,l,m+1,1))
                        + w[1]*(mc_0_r(2)*index(psi,l,m-1,0)+mc_0_r(3)*index(psi,l,m+1,0))
                        + w[2]*((-mc_0_r(4))*index(psi,l,m,1))
                        + lambda*gamm[0*3+0]*(mc_0_r(5)*index(psi,l-2,m-2,0)+mc_0_r(6)*index(psi,l-2,m,0)+mc_0_r(7)*index(psi,l-2,m+2,0)+mc_0_r(8)*index(psi,l,m-2,0)+mc_0_r(9)*index(psi,l,m,0)+mc_0_r(10)*index(psi,l,m+2,0)+mc_0_r(11)*index(psi,l+2,m-2,0)+mc_0_r(12)*index(psi,l+2,m,0)+mc_0_r(13)*index(psi,l+2,m+2,0))
                        + lambda*gamm[0*3+1]*((-mc_0_r(14))*index(psi,l-2,m-2,1)+(-mc_0_r(15))*index(psi,l-2,m+2,1)+(-mc_0_r(16))*index(psi,l,m-2,1)+(-mc_0_r(17))*index(psi,l,m+2,1)+(-mc_0_r(18))*index(psi,l+2,m-2,1)+(-mc_0_r(19))*index(psi,l+2,m+2,1))
                        + lambda*gamm[0*3+2]*(mc_0_r(20)*index(psi,l-2,m-1,0)+mc_0_r(21)*index(psi,l-2,m+1,0)+mc_0_r(22)*index(psi,l,m-1,0)+mc_0_r(23)*index(psi,l,m+1,0)+mc_0_r(24)*index(psi,l+2,m-1,0)+mc_0_r(25)*index(psi,l+2,m+1,0))
                        + lambda*gamm[1*3+1]*(mc_0_r(26)*index(psi,l-2,m-2,0)+mc_0_r(27)*index(psi,l-2,m,0)+mc_0_r(28)*index(psi,l-2,m+2,0)+mc_0_r(29)*index(psi,l,m-2,0)+mc_0_r(30)*index(psi,l,m,0)+mc_0_r(31)*index(psi,l,m+2,0)+mc_0_r(32)*index(psi,l+2,m-2,0)+mc_0_r(33)*index(psi,l+2,m,0)+mc_0_r(34)*index(psi,l+2,m+2,0))
                        + lambda*gamm[1*3+2]*((-mc_0_r(35))*index(psi,l-2,m-1,1)+(-mc_0_r(36))*index(psi,l-2,m+1,1)+(-mc_0_r(37))*index(psi,l,m-1,1)+(-mc_0_r(38))*index(psi,l,m+1,1)+(-mc_0_r(39))*index(psi,l+2,m-1,1)+(-mc_0_r(40))*index(psi,l+2,m+1,1))
                        + lambda*gamm[2*3+2]*(mc_0_r(41)*index(psi,l-2,m,0)+mc_0_r(42)*index(psi,l,m,0)+mc_0_r(43)*index(psi,l+2,m,0))
                        + Dr*(mc_0_r(44)*index(psi,l,m,0));
      psidot[1] = w[0]*(mc_0_r(0)*index(psi,l,m-1,0)+mc_0_r(1)*index(psi,l,m+1,0))
                        + w[1]*(mc_0_r(2)*index(psi,l,m-1,1)+mc_0_r(3)*index(psi,l,m+1,1))
                        + w[2]*(mc_0_r(4)*index(psi,l,m,0))
                        + lambda*gamm[0*3+0]*(mc_0_r(5)*index(psi,l-2,m-2,1)+mc_0_r(6)*index(psi,l-2,m,1)+mc_0_r(7)*index(psi,l-2,m+2,1)+mc_0_r(8)*index(psi,l,m-2,1)+mc_0_r(9)*index(psi,l,m,1)+mc_0_r(10)*index(psi,l,m+2,1)+mc_0_r(11)*index(psi,l+2,m-2,1)+mc_0_r(12)*index(psi,l+2,m,1)+mc_0_r(13)*index(psi,l+2,m+2,1))
                        + lambda*gamm[0*3+1]*(mc_0_r(14)*index(psi,l-2,m-2,0)+mc_0_r(15)*index(psi,l-2,m+2,0)+mc_0_r(16)*index(psi,l,m-2,0)+mc_0_r(17)*index(psi,l,m+2,0)+mc_0_r(18)*index(psi,l+2,m-2,0)+mc_0_r(19)*index(psi,l+2,m+2,0))
                        + lambda*gamm[0*3+2]*(mc_0_r(20)*index(psi,l-2,m-1,1)+mc_0_r(21)*index(psi,l-2,m+1,1)+mc_0_r(22)*index(psi,l,m-1,1)+mc_0_r(23)*index(psi,l,m+1,1)+mc_0_r(24)*index(psi,l+2,m-1,1)+mc_0_r(25)*index(psi,l+2,m+1,1))
                        + lambda*gamm[1*3+1]*(mc_0_r(26)*index(psi,l-2,m-2,1)+mc_0_r(27)*index(psi,l-2,m,1)+mc_0_r(28)*index(psi,l-2,m+2,1)+mc_0_r(29)*index(psi,l,m-2,1)+mc_0_r(30)*index(psi,l,m,1)+mc_0_r(31)*index(psi,l,m+2,1)+mc_0_r(32)*index(psi,l+2,m-2,1)+mc_0_r(33)*index(psi,l+2,m,1)+mc_0_r(34)*index(psi,l+2,m+2,1))
                        + lambda*gamm[1*3+2]*(mc_0_r(35)*index(psi,l-2,m-1,0)+mc_0_r(36)*index(psi,l-2,m+1,0)+mc_0_r(37)*index(psi,l,m-1,0)+mc_0_r(38)*index(psi,l,m+1,0)+mc_0_r(39)*index(psi,l+2,m-1,0)+mc_0_r(40)*index(psi,l+2,m+1,0))
                        + lambda*gamm[2*3+2]*(mc_0_r(41)*index(psi,l-2,m,1)+mc_0_r(42)*index(psi,l,m,1)+mc_0_r(43)*index(psi,l+2,m,1))
                        + Dr*(mc_0_r(44)*index(psi,l,m,1));
  }
  __syncthreads();
  if (l<=param->max_order && m<=l) {
    if (do_adams_bash_2) {
      olddiffx = promote_to_float2(psidot_io[ind(l,m,0)]);
      index(psi,l,m,0) += 3*h/2*psidot[0] - h/2*olddiffx.x;
      index(psi,l,m,1) += 3*h/2*psidot[1] - h/2*olddiffx.y;
      olddiffx =  promote_to_float2(psidot[0]);
      promote_to_float2(psi_in[ind(l,m,0)]) = promote_to_float2(index(psi,l,m,0));
    }
    promote_to_float2(psidot_io[ind(l,m,0)]) = promote_to_float2(psidot[0]);
  }
  __syncthreads();
}

void compute_psidot(REAL* psidot, REAL* psi, param_list_t *param, param_list_t *param_d, int do_adams_bash_2, int nr_times) {
  dim3 dimblock(block_width,block_width);
  dim3 dimgrid(param->data_width/block_width,param->data_width/block_width/2);
  int i;
  {
    if (first_mc_0) initialize_mc_0(param);
    if (first_thread_0) {
      first_thread_0 = 0;
      hipMalloc((void**)&mult_constant_0_d, sizeof(REAL)*param->data_width/2*param->data_width*mc_count_0);
      hipMemcpy(mult_constant_0_d,mult_constant_0,sizeof(REAL)*param->data_width/2*param->data_width*mc_count_0,hipMemcpyHostToDevice);
      hipBindTexture(0,mult_constant_0_r,mult_constant_0_d,sizeof(REAL)*param->data_width/2*param->data_width*mc_count_0);
    }

    for (i=0;i<nr_times;i++)
      cuda_thread_0<<<dimgrid,dimblock>>>(psidot, psi, param_d, mult_constant_0_d, do_adams_bash_2);
  }
}

static void initialize_mc_0(param_list_t *param) {
  int l,m;
  REAL ll,mm;

  first_mc_0 = 0;
  mult_constant_0 = (REAL*)malloc(sizeof(REAL)*param->data_width/2*param->data_width*mc_count_0);
  for (l=0;l<=param->max_order;l+=2) for (m=0;m<=l;m++) {
    ll = l;
    mm = m;
    if (abs(m-1)<=l)
      mc_0(0) = -(sqrt(1+ll-mm)*sqrt(ll+mm))/4.;
    else
      mc_0(0) = 0;
    if (abs(m+1)<=l)
      mc_0(1) = -(sqrt(ll-mm)*sqrt(1+ll+mm))/4.;
    else
      mc_0(1) = 0;
    if (abs(m-1)<=l)
      mc_0(2) = -(sqrt(1+ll-mm)*sqrt(ll+mm))/4.;
    else
      mc_0(2) = 0;
    if (abs(m+1)<=l)
      mc_0(3) = (sqrt(ll-mm)*sqrt(1+ll+mm))/4.;
    else
      mc_0(3) = 0;
    if (abs(m)<=l)
      mc_0(4) = -mm/2.;
    else
      mc_0(4) = 0;
    if (abs(m-2)<=l-2)
      mc_0(5) = ((1+ll)*sqrt(-3+ll+mm)*sqrt(-2+ll+mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(8.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(5) = 0;
    if (abs(m)<=l-2)
      mc_0(6) = -((1+ll)*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(4.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(6) = 0;
    if (abs(m+2)<=l-2)
      mc_0(7) = ((1+ll)*sqrt(-3+ll-mm)*sqrt(-2+ll-mm)*sqrt(-1+ll-mm)*sqrt(ll-mm))/(8.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(7) = 0;
    if (abs(m-2)<=l)
      mc_0(8) = (-3*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(8.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(8) = 0;
    if (abs(m)<=l)
      mc_0(9) = (ll+pow(ll,2)-3*pow(mm,2))/(12-16*ll-16*pow(ll,2));
    else
      mc_0(9) = 0;
    if (abs(m+2)<=l)
      mc_0(10) = (-3*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(8.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(10) = 0;
    if (abs(m-2)<=l+2)
      mc_0(11) = -((ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(3+ll-mm)*sqrt(4+ll-mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(24+16*ll)));
    else
      mc_0(11) = 0;
    if (abs(m)<=l+2)
      mc_0(12) = (ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(12+8*ll));
    else
      mc_0(12) = 0;
    if (abs(m+2)<=l+2)
      mc_0(13) = -((ll*sqrt(1+ll+mm)*sqrt(2+ll+mm)*sqrt(3+ll+mm)*sqrt(4+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(24+16*ll)));
    else
      mc_0(13) = 0;
    if (abs(m-2)<=l-2)
      mc_0(14) = -((1+ll)*sqrt(-3+ll+mm)*sqrt(-2+ll+mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(4.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(14) = 0;
    if (abs(m+2)<=l-2)
      mc_0(15) = ((1+ll)*sqrt(-3+ll-mm)*sqrt(-2+ll-mm)*sqrt(-1+ll-mm)*sqrt(ll-mm))/(4.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(15) = 0;
    if (abs(m-2)<=l)
      mc_0(16) = (3*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(16) = 0;
    if (abs(m+2)<=l)
      mc_0(17) = (-3*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(17) = 0;
    if (abs(m-2)<=l+2)
      mc_0(18) = (ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(3+ll-mm)*sqrt(4+ll-mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(12+8*ll));
    else
      mc_0(18) = 0;
    if (abs(m+2)<=l+2)
      mc_0(19) = -((ll*sqrt(1+ll+mm)*sqrt(2+ll+mm)*sqrt(3+ll+mm)*sqrt(4+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(12+8*ll)));
    else
      mc_0(19) = 0;
    if (abs(m-1)<=l-2)
      mc_0(20) = -((1+ll)*sqrt(ll-mm)*sqrt(-2+ll+mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(2.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(20) = 0;
    if (abs(m+1)<=l-2)
      mc_0(21) = ((1+ll)*sqrt(-2+ll-mm)*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(ll+mm))/(2.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(21) = 0;
    if (abs(m-1)<=l)
      mc_0(22) = (3*(1-2*mm)*sqrt(1+ll-mm)*sqrt(ll+mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(22) = 0;
    if (abs(m+1)<=l)
      mc_0(23) = (-3*sqrt(ll-mm)*sqrt(1+ll+mm)*(1+2*mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(23) = 0;
    if (abs(m-1)<=l+2)
      mc_0(24) = -((ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(3+ll-mm)*sqrt(1+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(6+4*ll)));
    else
      mc_0(24) = 0;
    if (abs(m+1)<=l+2)
      mc_0(25) = (ll*sqrt(1+ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm)*sqrt(3+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(6+4*ll));
    else
      mc_0(25) = 0;
    if (abs(m-2)<=l-2)
      mc_0(26) = -((1+ll)*sqrt(-3+ll+mm)*sqrt(-2+ll+mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(8.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(26) = 0;
    if (abs(m)<=l-2)
      mc_0(27) = -((1+ll)*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(4.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(27) = 0;
    if (abs(m+2)<=l-2)
      mc_0(28) = -((1+ll)*sqrt(-3+ll-mm)*sqrt(-2+ll-mm)*sqrt(-1+ll-mm)*sqrt(ll-mm))/(8.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(28) = 0;
    if (abs(m-2)<=l)
      mc_0(29) = (3*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(8.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(29) = 0;
    if (abs(m)<=l)
      mc_0(30) = (ll+pow(ll,2)-3*pow(mm,2))/(12-16*ll-16*pow(ll,2));
    else
      mc_0(30) = 0;
    if (abs(m+2)<=l)
      mc_0(31) = (3*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(8.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(31) = 0;
    if (abs(m-2)<=l+2)
      mc_0(32) = (ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(3+ll-mm)*sqrt(4+ll-mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(24+16*ll));
    else
      mc_0(32) = 0;
    if (abs(m)<=l+2)
      mc_0(33) = (ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(12+8*ll));
    else
      mc_0(33) = 0;
    if (abs(m+2)<=l+2)
      mc_0(34) = (ll*sqrt(1+ll+mm)*sqrt(2+ll+mm)*sqrt(3+ll+mm)*sqrt(4+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(24+16*ll));
    else
      mc_0(34) = 0;
    if (abs(m-1)<=l-2)
      mc_0(35) = ((1+ll)*sqrt(ll-mm)*sqrt(-2+ll+mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(2.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(35) = 0;
    if (abs(m+1)<=l-2)
      mc_0(36) = ((1+ll)*sqrt(-2+ll-mm)*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(ll+mm))/(2.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(36) = 0;
    if (abs(m-1)<=l)
      mc_0(37) = (3*sqrt(1+ll-mm)*sqrt(ll+mm)*(-1+2*mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(37) = 0;
    if (abs(m+1)<=l)
      mc_0(38) = (-3*sqrt(ll-mm)*sqrt(1+ll+mm)*(1+2*mm))/(4.*(-3+4*ll+4*pow(ll,2)));
    else
      mc_0(38) = 0;
    if (abs(m-1)<=l+2)
      mc_0(39) = (ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(3+ll-mm)*sqrt(1+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(6+4*ll));
    else
      mc_0(39) = 0;
    if (abs(m+1)<=l+2)
      mc_0(40) = (ll*sqrt(1+ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm)*sqrt(3+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(6+4*ll));
    else
      mc_0(40) = 0;
    if (abs(m)<=l-2)
      mc_0(41) = ((1+ll)*sqrt(-1+ll-mm)*sqrt(ll-mm)*sqrt(-1+ll+mm)*sqrt(ll+mm))/(2.*sqrt(-3+2*ll)*(-1+2*ll)*sqrt(1+2*ll));
    else
      mc_0(41) = 0;
    if (abs(m)<=l)
      mc_0(42) = (ll+pow(ll,2)-3*pow(mm,2))/(-6+8*ll+8*pow(ll,2));
    else
      mc_0(42) = 0;
    if (abs(m)<=l+2)
      mc_0(43) = -((ll*sqrt(1+ll-mm)*sqrt(2+ll-mm)*sqrt(1+ll+mm)*sqrt(2+ll+mm))/(sqrt(1+2*ll)*sqrt(5+2*ll)*(6+4*ll)));
    else
      mc_0(43) = 0;
    if (abs(m)<=l)
      mc_0(44) = -(ll*(1+ll));
    else
      mc_0(44) = 0;
  }
}
